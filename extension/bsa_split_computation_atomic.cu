#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#define D 64
#define NUM_CHUNKS 4
constexpr int BLOCK_SIZE = 16;

__device__ __forceinline__ float atomicMaxFloat (float * addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMax((int *)addr, __float_as_int(value))) :
         __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));

    return old;
}

__global__
void forward_kernel(
    const float* Q,
    const float* K,
    const float* V,
    const int* block_indices,
    const int num_blocks_selected,
    const int num_blocks,
    float* output,
    const int T
) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int b = blockIdx.y;

    float P[BLOCK_SIZE];
    float acc[D] = {0};
    __shared__ float shared_acc[BLOCK_SIZE][D];
    __shared__ float shared_max[BLOCK_SIZE];
    __shared__ float shared_sum[BLOCK_SIZE];

    // if(ty == 0){
    for(int d = (D / NUM_CHUNKS) * ty; d < (D / NUM_CHUNKS) * (ty + 1); d++){
        shared_acc[tx][d] = 0;
    }

    if(ty == 0){
        shared_max[tx] = -INFINITY;
        shared_sum[tx] = 0;
    }

    float sum = 0;
    float curr_max = -INFINITY;

    int q_idx = (b * T + bx * BLOCK_SIZE + tx) * D;

    for(int i = ty * (num_blocks_selected / NUM_CHUNKS); i < (ty + 1) * (num_blocks_selected / NUM_CHUNKS); i++){
        int block = block_indices[(b * num_blocks + bx) * num_blocks_selected + i];
        float new_max = curr_max;
        for(int j = 0; j < BLOCK_SIZE; j++){
            float weight = 0;
            for(int d = 0; d < D; d++){
                weight += Q[q_idx + d] * K[(b * T * D) + (block * BLOCK_SIZE * D) + (j * D) + d];
            }
            new_max = fmaxf(new_max, weight);
            P[j] = weight;
        }

        float difference = expf(curr_max - new_max);
        sum *= difference;
        for(int d = 0; d < D; d++){
            acc[d] *= difference;
        }

        for(int j = 0; j < BLOCK_SIZE; j++){
            float norm_weight = expf(P[j] - new_max);
            for(int d = 0; d < D; d++){
                acc[d] += norm_weight * V[(b * T * D) + (block * BLOCK_SIZE * D) + (j * D) + d];
            }
            sum += norm_weight;
        }

        curr_max = new_max;
    }

    atomicMaxFloat(&shared_max[tx], curr_max);
    __syncthreads();


    for(int d = 0; d < D; d++){
        int sd = (d + ty * (D / NUM_CHUNKS)) % D;
        atomicAdd(&shared_acc[tx][sd], acc[sd] * expf(curr_max - shared_max[tx]));
    }
    atomicAdd(&shared_sum[tx], sum * expf(curr_max - shared_max[tx])); // how do I monkey patch everything?

    __syncthreads();

    // if(ty == 0){
    int out_idx = (b * T + bx * BLOCK_SIZE + tx) * D;
    for(int d = (ty) * (D / NUM_CHUNKS); d < D; d++){
        output[out_idx + d] = shared_acc[tx][d] / shared_sum[tx];
    }
    // }
}

torch::Tensor forward(
    torch::Tensor queries,
    torch::Tensor keys,
    torch::Tensor values,
    torch::Tensor query_blocks
) {
    int B = queries.size(0);
    int T = queries.size(1);
    // D should match the macro D
    int num_blocks_selected = query_blocks.size(2);
    int num_blocks = query_blocks.size(1);

    dim3 gridDim((T + BLOCK_SIZE - 1) / BLOCK_SIZE, B);
    dim3 blockDim(BLOCK_SIZE, NUM_CHUNKS);

    auto output = torch::zeros_like(queries);

    float* Q = queries.data_ptr<float>();
    float* K = keys.data_ptr<float>();
    float* V = values.data_ptr<float>();
    int* QB_ptr = query_blocks.data_ptr<int>();
    float* O = output.data_ptr<float>();

    forward_kernel<<<gridDim, blockDim>>>(Q, K, V, QB_ptr, num_blocks_selected, num_blocks, O, T);

    return output;
}