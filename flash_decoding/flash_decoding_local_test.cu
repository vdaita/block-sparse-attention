#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "flash_decoding.cuh"
#include <cmath>
#include <random>
#include <stdio.h>
#include <math.h>
#include <ctype.h>
#include <errno.h>

// Claude
float getNextFloat() {
    float value;
    if (scanf("%f", &value) == 1) {
        return value;
    }
    // Handle error or end of input
    return 0.0; // Or use a special value to indicate error
}

int main(int argc, char** argv){
    int B = 1;
    int T = 32;

    float* q = (float*) malloc(B * D * sizeof(float));
    float* k = (float*) malloc(B * T * D * sizeof(float));
    float* v = (float*) malloc(B * T * D * sizeof(float));

    float* device_q; 
    hipMalloc((void**) &device_q, B * D * sizeof(float));
    float* device_k; 
    hipMalloc((void**) &device_k, B * T * D * sizeof(float));
    float* device_v; 
    hipMalloc((void**) &device_v, B * T * D * sizeof(float));

    for(int i = 0; i < B * D; i++){
        q[i] = getNextFloat();
    }

    for(int i = 0; i < B * T * D; i++){
        k[i] = getNextFloat();
    }

    for(int i = 0; i < B * T * D; i++){
        v[i] = getNextFloat();
    }

    float* target_output = (float*) malloc(B * D * sizeof(float));
    for(int i = 0; i < B * D; i++){
        target_output[i] = getNextFloat();
    }

    // std::mt19937 gen(42);
    // std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

    // for(int i = 0; i < B * D; i++){
    //     q[i] = dist(gen);
    // }

    // for(int i = 0; i < B * T * D; i++){
    //     k[i] = dist(gen);
    //     v[i] = dist(gen);
    // }

    hipMemcpy(device_q, q, B * D * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_k, k, B * T * D * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_v, v, B * T * D * sizeof(float), hipMemcpyHostToDevice);

    float* device_o;
    hipMalloc((void**) &device_o, B * D * sizeof(float)); 
    float* o = (float*) malloc(B * D * sizeof(float));

    int num_blocks_for_head = min((T + BLOCK_TOKENS - 1) / BLOCK_TOKENS, 8);
    dim3 gridDim(1, num_blocks_for_head, B);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_TOKENS, 1);

    shared_split_k_kernel<<<gridDim, blockDim>>>(
        device_q,
        device_k,
        device_v,
        device_o,
        B,
        T
    );

    hipMemcpy(o, device_o, B * D * sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 0; i < D; i++){
        printf("%f ", o[i]);
    }
    printf("\n");
}