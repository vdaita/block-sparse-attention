#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "flash_decoding.cuh"
#include <cmath>
#include <random>
#include <stdio.h>
#include <math.h>
#include <ctype.h>
#include <errno.h>

// GPT
void checkLastCudaError() {
    hipError_t error = hipGetLastError(); // Get the last error
    if (error != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(error)); // Print the error string
    } else {
        printf("No CUDA error.\n");
    }
}


// Claude
float getNextFloat() {
    float value;
    if (scanf("%f", &value) == 1) {
        return value;
    }
    // Handle error or end of input
    return 0.0; // Or use a special value to indicate error
}

int main(int argc, char** argv){
    int B = 1;
    int T = 64;

    float* q = (float*) malloc(B * D * sizeof(float));
    float* k = (float*) malloc(B * T * D * sizeof(float));
    float* v = (float*) malloc(B * T * D * sizeof(float));

    float* device_q; 
    hipMalloc((void**) &device_q, B * D * sizeof(float));
    float* device_k; 
    hipMalloc((void**) &device_k, B * T * D * sizeof(float));
    float* device_v; 
    hipMalloc((void**) &device_v, B * T * D * sizeof(float));

    for(int i = 0; i < B * D; i++){
        q[i] = getNextFloat();
    }

    for(int i = 0; i < B * T * D; i++){
        k[i] = getNextFloat();
    }

    for(int i = 0; i < B * T * D; i++){
        v[i] = getNextFloat();
    }

    float* target_output = (float*) malloc(B * D * sizeof(float));
    for(int i = 0; i < B * D; i++){
        target_output[i] = getNextFloat();
    }

    // std::mt19937 gen(42);
    // std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

    // for(int i = 0; i < B * D; i++){
    //     q[i] = dist(gen);
    // }

    // for(int i = 0; i < B * T * D; i++){
    //     k[i] = dist(gen);
    //     v[i] = dist(gen);
    // }

    hipMemcpy(device_q, q, B * D * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_k, k, B * T * D * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_v, v, B * T * D * sizeof(float), hipMemcpyHostToDevice);

    int num_blocks_per_head = min((T + BLOCK_TOKENS - 1) / BLOCK_TOKENS, 1);
    dim3 gridDim(1, num_blocks_per_head, B);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_TOKENS, 1);

    float* device_o;
    hipMalloc((void**) &device_o, B * D * num_blocks_per_head * sizeof(float)); 
    float* o = (float*) malloc(B * D * num_blocks_per_head * sizeof(float));

    float* device_o_sum;
    hipMalloc((void**) &device_o_sum, B * num_blocks_per_head * sizeof(float));
    float* device_o_max;
    hipMalloc((void**) &device_o_max, B * num_blocks_per_head * sizeof(float));

    float* o_sum = (float*) malloc(B * num_blocks_per_head * sizeof(float));
    float* o_max = (float*) malloc(B * num_blocks_per_head * sizeof(float));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Shared memory per block: %d bytes\n", prop.sharedMemPerBlock);
    printf("Num blocks per head %d\n", num_blocks_per_head);

    shared_split_k_kernel<<<gridDim, blockDim>>>(
        device_q,
        device_k,
        device_v,
        device_o,
        device_o_sum,
        device_o_max,
        B,
        T
    );

    checkLastCudaError();

    hipMemcpy(o, device_o, B * D * num_blocks_per_head * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(o_sum, device_o_sum, B * num_blocks_per_head * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(o_max, device_o_max, B * num_blocks_per_head * sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 0; i < D; i++){
        printf("%f ", (o[i] / o_sum[0]));
    }
    printf("\n");

    bool works = true;
    for(int i = 0; i < D; i++){
        if(abs((o[i] / o_sum[0]) - target_output[i]) > 0.02){
            works = false;
            break;
        }
    }

    if(works){
        printf("Works!\n");
    } else {
        printf("Doesn't work!\n");
    }
}